#include "hip/hip_runtime.h"
#include <iostream>
#include <cu_exp_statistics.hpp>
#include <cutf/experimental/fp.hpp>
#include <cutf/memory.hpp>

namespace {
using count_t = unsigned long long int;

template <class T>
__global__ void statistics_kernel(
		count_t* const result_ptr,
		const T* const ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t ld
		) {
	const std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= m * n) {
		return;
	}

	const auto im = tid % m;
	const auto in = tid / m;

	const auto mem_index = im + in * ld;
	const auto value = ptr[mem_index];

	if ((cutf::experimental::fp::reinterpret_as_uint(value) << 1) == 0) {
		atomicAdd(result_ptr, 1lu);
		return;
	}

	const auto exp = cutf::experimental::fp::mask_exponent(value) >> cutf::experimental::fp::get_mantissa_size<T>();

	atomicAdd(result_ptr + exp + 1, 1lu);
}

__global__ void init_array_kernel(
		count_t* const ptr,
		const std::size_t size
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= size) {
		return;
	}

	ptr[tid] = 0;
}
} // unnamed namespace

template <class T>
mtk::cu_exp_statistics::result_t mtk::cu_exp_statistics::take_vector_statistics(
		const T* const ptr,
		const std::size_t size,
		hipStream_t hip_stream
		) {
	return mtk::cu_exp_statistics::take_matrix_statistics(
			ptr,
			size,
			1,
			1
			);
}

template <class T>
mtk::cu_exp_statistics::result_t mtk::cu_exp_statistics::take_matrix_statistics(
		const T* const ptr,
		const std::size_t m,
		const std::size_t n,
		const std::size_t ld_,
		hipStream_t hip_stream
		) {
	const std::size_t ld = ld_ == 0 ? m : ld_;

	const std::size_t statistics_array_size = (1lu << cutf::experimental::fp::get_exponent_size<T>()) + 1;
	count_t *dev_count;
	count_t *hos_count;
	CUTF_CHECK_ERROR(hipMalloc(&dev_count, sizeof(count_t) * statistics_array_size));
	CUTF_CHECK_ERROR(hipHostMalloc(&hos_count, sizeof(count_t) * statistics_array_size));

	const std::size_t size = m * n;
	const auto block_size = 256;
	const auto grid_size = (size + block_size - 1) / block_size;

	init_array_kernel<<<grid_size, block_size, 0, hip_stream>>>(dev_count, statistics_array_size);
	statistics_kernel<<<grid_size, block_size, 0, hip_stream>>>(dev_count, ptr, m, n, ld);

	CUTF_CHECK_ERROR(hipMemcpyAsync(hos_count, dev_count, sizeof(count_t) * statistics_array_size, hipMemcpyDefault, hip_stream));
	CUTF_CHECK_ERROR(hipStreamSynchronize(hip_stream));

	mtk::cu_exp_statistics::result_t result;
	result.num_zero = hos_count[0];

	for (std::uint32_t i = 0; i < (1u << cutf::experimental::fp::get_exponent_size<T>()); i++) {
		if (hos_count[i + 1] != 0) {
			result.distribution.insert(std::make_pair(
						static_cast<int>(i) - cutf::experimental::fp::get_bias<T>(),
						hos_count[i + 1]
						));
		}
	}

	CUTF_CHECK_ERROR(hipHostFree(hos_count));
	CUTF_CHECK_ERROR(hipFree(dev_count));

	return result;
}

std::string mtk::cu_exp_statistics::to_json(
		const mtk::cu_exp_statistics::result_t& result
		) {
	std::string str = "{";
	str += "num_zero:" + std::to_string(result.num_zero);

	for (int exp = -10000; exp <= 10000; exp++) {
		if (result.distribution.count(exp) != 0) {
			str += ",\"" + std::to_string(exp) + "\":" + std::to_string(result.distribution.at(exp));
		}
	}
	str += "}";

	return str;
}

#define TAKE_MATRIX_STATISTICS_INSTANCE(type)\
	template mtk::cu_exp_statistics::result_t mtk::cu_exp_statistics::take_matrix_statistics<type>( \
		const type* const ptr, \
		const std::size_t m, \
		const std::size_t n, \
		const std::size_t ld, \
		hipStream_t hip_stream \
		)
#define TAKE_VECTOR_STATISTICS_INSTANCE(type)\
	template mtk::cu_exp_statistics::result_t mtk::cu_exp_statistics::take_vector_statistics<type>( \
		const type* const ptr, \
		const std::size_t size, \
		hipStream_t hip_stream \
		)

TAKE_MATRIX_STATISTICS_INSTANCE(half  );
TAKE_MATRIX_STATISTICS_INSTANCE(float );
TAKE_MATRIX_STATISTICS_INSTANCE(double);
TAKE_VECTOR_STATISTICS_INSTANCE(half  );
TAKE_VECTOR_STATISTICS_INSTANCE(float );
TAKE_VECTOR_STATISTICS_INSTANCE(double);
