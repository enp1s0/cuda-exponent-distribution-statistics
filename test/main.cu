#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cu_exp_statistics.hpp>

template <class T>
struct base_t {
	using type = T;
	static const unsigned num_elements = 1;
};
template <> struct base_t<double2> {using type = double;static const unsigned num_elements = 2;};
template <> struct base_t<float2 > {using type = float; static const unsigned num_elements = 2;};

template <class T>
void test(const std::size_t m, const std::size_t n) {
	const auto size = m * n * base_t<T>::num_elements;
	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<typename base_t<T>::type> dist(-10, 10);

	T* test_matrix_ptr;
	hipMallocManaged(&test_matrix_ptr, sizeof(T) * size);
	for (std::size_t i = 0; i < size; i++) {
		reinterpret_cast<typename base_t<T>::type*>(test_matrix_ptr)[i] = dist(mt);
	}
	hipDeviceSynchronize();

	const auto result = mtk::cu_exp_statistics::take_matrix_statistics(
			test_matrix_ptr,
			m, n
			);
	std::printf("# zero = %lu\n", result.num_zero);
	std::printf("# inf = %lu\n", result.num_inf);
	std::printf("# nan = %lu\n", result.num_nan);
	for (int exp = -2048; exp <= 2048; exp++) {
		if (result.distribution.count(exp) != 0) {
			std::size_t count = result.distribution.at(exp);
			std::printf("[%+4d]: %lu\n", exp, count);
		}
	}

	std::printf("JSON: %s\n", mtk::cu_exp_statistics::to_json(result).c_str());

	hipFree(test_matrix_ptr);
}

int main() {
	test<float  >(20000, 20000);
	test<double >(20000, 20000);
	test<float2 >(20000, 20000);
	test<double2>(20000, 20000);
}
